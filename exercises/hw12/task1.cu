
#include <hip/hip_runtime.h>
#include <iostream>
// Thread block size
#define BLOCK_SIZE 32

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
  int width;
  int height;
  int stride;
  float *elements;
} Matrix;

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col) {
  return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col, float value) {
  A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
__device__ Matrix GetSubMatrix(Matrix A, int row, int col) {
  Matrix Asub;
  Asub.width = BLOCK_SIZE;
  Asub.height = BLOCK_SIZE;
  Asub.stride = A.stride;
  Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
  return Asub;
}

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C) {
  // Load A and B to device memory
  Matrix d_A;
  d_A.width = d_A.stride = A.width;
  d_A.height = A.height;
  size_t size = A.width * A.height * sizeof(float);
  hipMalloc(&d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
  Matrix d_B;
  d_B.width = d_B.stride = B.width;
  d_B.height = B.height;
  size = B.width * B.height * sizeof(float);
  hipMalloc(&d_B.elements, size);
  hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

  // Allocate C in device memory
  Matrix d_C;
  d_C.width = d_C.stride = C.width;
  d_C.height = C.height;
  size = C.width * C.height * sizeof(float);
  hipMalloc(&d_C.elements, size);

  // Invoke kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
  MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

  // Read C from device memory
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
  // Block row and column
  int blockRow = blockIdx.y;
  int blockCol = blockIdx.x;

  // Each thread block computes one sub-matrix Csub of C
  Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

  // Each thread computes one element of Csub
  // by accumulating results into Cvalue
  float Cvalue = 0;

  // Thread row and column within Csub
  int row = threadIdx.y;
  int col = threadIdx.x;

  // Loop over all the sub-matrices of A and B that are
  // required to compute Csub
  // Multiply each pair of sub-matrices together
  // and accumulate the results
  for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

    // Get sub-matrix Asub of A
    Matrix Asub = GetSubMatrix(A, blockRow, m);

    // Get sub-matrix Bsub of B
    Matrix Bsub = GetSubMatrix(B, m, blockCol);

    // Shared memory used to store Asub and Bsub respectively
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load Asub and Bsub from device memory to shared memory
    // Each thread loads one element of each sub-matrix
    As[row][col] = GetElement(Asub, row, col);
    Bs[row][col] = GetElement(Bsub, row, col);

    // Synchronize to make sure the sub-matrices are loaded
    // before starting the computation
    __syncthreads();
    // Multiply Asub and Bsub together
    // Original erroneous code: for (int e = 0; e <= BLOCK_SIZE; ++e)
    for (int e = 0; e < BLOCK_SIZE; ++e)
      Cvalue += As[row][e] * Bs[e][col];
    __syncthreads(); // Additional sync to fix shared mem race conditions
  }

  // Write Csub to device memory
  // Each thread writes one element
  SetElement(Csub, row, col, Cvalue);
}

int main() {
  const int num_m = 3;           // we need 3 matrices
  const int side_dim = 128;      // side dimension of square matrix
  Matrix *m = new Matrix[num_m]; // allocate matrix storage part 1
  for (int i = 0; i < num_m; i++) {
    m[i].width = m[i].height = m[i].stride = side_dim; // set matrix params
    m[i].elements =
        new float[side_dim * side_dim]; // allocate matrix storage part 2
    if (i < 2)                          // initialize first two matrices
      for (int j = 0; j < side_dim * side_dim; j++)
        m[i].elements[j] = 1.0f;
  }
  MatMul(m[0], m[1], m[2]); // perform matrix-multiply
  std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
  for (int i = 0; i < side_dim * side_dim; i++) // perform results checking
    if (m[2].elements[i] != (float)side_dim) {
      std::cout << "Mismatch at index: " << i
                << " expected: " << (float)side_dim
                << " got: " << m[2].elements[i] << std::endl;
      return 0;
    }
  std::cout << "Success!" << std::endl;
  for (int i = 0; i < num_m; i++)
    delete[] m[i].elements;
  delete[] m;
  return 0;
}
