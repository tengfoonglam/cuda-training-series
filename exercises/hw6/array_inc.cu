#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
// error checking macro
#define cudaCheckErrors(msg)                                                   \
  do {                                                                         \
    hipError_t __err = hipGetLastError();                                    \
    if (__err != hipSuccess) {                                                \
      fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", msg,                  \
              hipGetErrorString(__err), __FILE__, __LINE__);                  \
      fprintf(stderr, "*** FAILED - ABORTING\n");                              \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

template <typename T> void alloc_bytes(T &ptr, size_t num_bytes) {

  hipMallocManaged(&ptr, num_bytes);
}

__global__ void inc(int *array, size_t n) {
  size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
  while (idx < n) {
    ++array[idx];
    idx += blockDim.x * gridDim.x; // grid-stride loop
  }
}

const size_t ds = 32ULL * 1024ULL * 1024ULL;

int main() {

  int *um_array;
  alloc_bytes(um_array, ds * sizeof(um_array[0]));
  cudaCheckErrors("hipMalloc Error");
  memset(um_array, 0, ds * sizeof(um_array[0]));
  hipMemPrefetchAsync(um_array, ds * sizeof(um_array[0]), 0);
  inc<<<256, 256>>>(um_array, ds);
  cudaCheckErrors("kernel launch error");
  hipMemPrefetchAsync(um_array, ds * sizeof(um_array[0]), hipCpuDeviceId);
  hipDeviceSynchronize();
  cudaCheckErrors("hipDeviceSynchronize Error");
  for (int i = 0; i < ds; i++)
    if (um_array[i] != 1) {
      printf("mismatch at %d, was: %d, expected: %d\n", i, um_array[i], 1);
      return -1;
    }
  printf("success!\n");
  return 0;
}
